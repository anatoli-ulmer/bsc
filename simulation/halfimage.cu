
#include <hip/hip_runtime.h>
#ifdef wip
__global__ void halfy(const int N, const double *data, double *out)
{
#ifdef shared
    extern  __shared__ double in[];
#endif
    //�ber y=1:n bei festem x aus 1..2N
    
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    
    if (x<2*N&&y<N) {
#ifdef shared
        
        int step=min(N,blockDim.y);
        int inpos=x*2*N + y;
        int sharepos=threadIdx.y*2+1;
        
        if (threadIdx.y==0)in[0]=((blockIdx.y>0)?data[inpos-1]:0);
        in[1+threadIdx.y]=data[inpos];
        in[1+threadIdx.y+step]=data[inpos+step];
        __syncthreads();
        
        double prev=in[sharepos-1];
        double center=in[sharepos];
        double next=in[sharepos+1];
        
#else
        int inpos=x*2*N + 2 * y;
        double prev= y>0?data[inpos-1]:0;
        double center = data[inpos];
        double next = data[inpos+1];
#endif
        
        out[x*N + y] =(prev+2*center+next) /((y>0)?4:3);
    }
}


__global__ void halfx(const int N, const double *data, double *out)
{
    
    //�ber x=1:n bei festem y aus 1..n
    
    
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    
    
    
	if (x<N&&y<N) {
    double prev= x>0?data[(x*2-1)*N + y]:0;
    double center = data[x*2*N + y];
    double next = data[(x*2+1)*N +  y];
    out[x*N + y] = (prev+2*center+next)/((x>0)+3);
//         out[x*N + y]=prev;
 	}
}
#endif
__global__ void halfimage(const int N, const double *data, double *out)
{
    const int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    if (x<N&&y<N) {
        const int N2=2*N;
        const int x2=x*2;
        const int y2=y*2;
        double left=0;
        if (x>0){
            left=(2*data[(x2-1)*N2+y2]+data[(x2-1)*N2+y2+1]);
            if (y>0)left+=data[(x2-1)*N2+y2-1];
        }
        if (y>0)left=left/4;else left=left/3;
        
        double   center=(2*data[x2*N2+y2]+data[x2*N2+y2+1]);
        if (y>0) center+=data[x2*N2+y2-1];
        if (y>0) center=center/4; else center=center/3;
        
        double   right=(2*data[(x2+1)*N2+y2]+data[(x2+1)*N2+y2+1]);
        if (y>0) right+=data[(x2+1)*N2+y2-1];
        if (y>0) right=right/4;else right=right/3;
        
        out[x*N + y]=(left+2*center+right)/((x>0)?4:3);
    }
}

#include <hip/hip_runtime.h>
#ifdef wip
__global__ void conv_col(const int N, const double *data, double *out)
{
    //�ber y=1:n bei festem x aus 1..2N
    
    
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int inpos=x*N + 2*y;
    int sharepos=threadIdx.y*2+1;
//     const int size = blockDim.y*2+1;
    if (x<2*N&&y<N) {
#ifdef shared
        extern  __shared__ double in[];
        if (y==0)in[0]=((blockIdx.y>0)?data[inpos-1]:0);
        in[sharepos]=data[ inpos];
        in[sharepos+1]=data[ inpos+1];
        __syncthreads();
        
        double prev=in[sharepos-1];
        double center=in[sharepos];
        double next=in[sharepos+1];
        
#else
        double prev= y>0?data[x*N + 2 * y-1]:0;
        double center = data[ x*N + 2*y];
        double next = data[x*N + 2 * y];
#endif
        
        
        
        out[x*N + y] =(prev+2*center+next) /((y>0)+3);
        
    }
}


__global__ void conv_row(const int N, const double *data, double *out)
{
    
    //�ber x=1:n bei festem y aus 1..n
    
    
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    
    
    
// 	if (x<N&&y<N) {
    double prev= x>0?data[(x*2-1)*N + y]:0;
    double center = data[x*2*N + y];
    double next = data[(x*2-1)*N +  y];
    out[x*N + y] = (prev+2*center+next)/((x>0)+3);
//         out[x*N + y]=center;
// 	}
}
#endif
__global__ void halfimage(const int N, const double *data, double *out)
{
    const int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    if (x<N&&y<N) {
        const int N2=2*N;
        const int x2=x*2;
        const int y2=y*2;
        double left=0;
        if (x>0){
            left=(2*data[(x2-1)*N2+y2]+data[(x2-1)*N2+y2+1]);
            if (y>0)left+=data[(x2-1)*N2+y2-1];
        }
        if (y>0)left=left/4;else left=left/3;
        
        double   center=(2*data[x2*N2+y2]+data[x2*N2+y2+1]);
        if (y>0) center+=data[x2*N2+y2-1];
        if (y>0) center=center/4; else center=center/3;
        
        double   right=(2*data[(x2+1)*N2+y2]+data[(x2+1)*N2+y2+1]);
        if (y>0) right+=data[(x2+1)*N2+y2-1];
        if (y>0) right=right/4;else right=right/3;
        
        out[x*N + y]=(left+2*center+right)/((x>0)?4:3);
    }
}

#include <hip/hip_runtime.h>
__global__ void inpoly(const int N, const float *vertx, const float *verty,const int nv, bool *out )
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    if (x<N&&y<N){
        bool inpoly=false;
        
        for (int i = 0, j = nv-1; i < nv; j = i++) {
            if ( ((verty[i]>y) != (verty[j]>y)) &&
                    //inside verty range
                    (x < (vertx[j]-vertx[i]) * (y-verty[i]) / (verty[j]-verty[i]) + vertx[i]) )
            {
                inpoly = !inpoly;
            }
        }
        out[x*N+y]=inpoly;
    }
}

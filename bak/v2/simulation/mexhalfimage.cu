#include "hip/hip_runtime.h"
#include "mex.h"
#include "gpu/mxGPUArray.h"

/*
 * Device code
 */

__global__ void halfimage( int const N2, double const * const data, double * const out)
{
    const int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    const int x2=x*2;
    const int y2=y*2;
    
    if (x2<N2&&y2<N2) {
        double left=0;
        if (x>0){
            left=(2*data[(x2-1)*N2+y2]+data[(x2-1)*N2+y2+1]);
            if (y>0)left+=data[(x2-1)*N2+y2-1];
        }
        if (y>0)left=left/4;else left=left/3;
        
        double   center=(2*data[x2*N2+y2]+data[x2*N2+y2+1]);
        if (y>0) center+=data[x2*N2+y2-1];
        if (y>0) center=center/4; else center=center/3;
        
        double   right=(2*data[(x2+1)*N2+y2]+data[(x2+1)*N2+y2+1]);
        if (y>0) right+=data[(x2+1)*N2+y2-1];
        if (y>0) right=right/4;else right=right/3;
        
        out[x*N2/2 + y]=(left+2*center+right)/((x>0)?4:3);
    }
}

/*
 * Host code
 */
void mexFunction(int nlhs, mxArray *plhs[],
        int nrhs, mxArray const *prhs[])
{
    mxGPUArray const *A;
    mxGPUArray *B;
    double const *d_A;
    double *d_B;
    
    mxInitGPU();
    
    if ((nrhs!=1) || (nlhs!=1)) {
        mexErrMsgTxt("%d Expected one input and one output");
    }
    
    A = mxGPUCreateFromMxArray(prhs[0]);
    
    if (mxGPUGetClassID(A) != mxDOUBLE_CLASS) {
        mexErrMsgTxt("input must be double");
    }
    
    d_A = (double const *)(mxGPUGetDataReadOnly(A));
    int N = (int)(mxGPUGetNumberOfElements(A));
    
    int  ndims=mxGPUGetNumberOfDimensions(A);
    mwSize dimso[2]={0,0};
    switch (ndims) {
        case 1:
            dimso[0] = N/2;
            break;
        case 2:
            dimso[0]=mxGPUGetDimensions(A)[0]/2;
            dimso[1]=mxGPUGetDimensions(A)[1]/2;
            break;
        default:
            mexErrMsgTxt("input be 2d");
    }
    
    B = mxGPUCreateGPUArray(ndims,
            dimso,
            mxGPUGetClassID(A),
            mxGPUGetComplexity(A),
            MX_GPU_DO_NOT_INITIALIZE);
    d_B = (double *)(mxGPUGetData(B));
    
    int dimN=(int)floor(sqrt((double)N));
    dim3 threadsPerBlock(8, 8);
    dim3 numBlocks((int)ceil((double)(dimN/2)/ threadsPerBlock.x), (int)ceil((double)(dimN/2) / threadsPerBlock.y));
    
    halfimage<<<numBlocks, threadsPerBlock>>>(dimN,d_A, d_B);
    
    plhs[0] = mxGPUCreateMxArrayOnGPU(B);
    
    mxGPUDestroyGPUArray(A);
    mxGPUDestroyGPUArray(B);
}
